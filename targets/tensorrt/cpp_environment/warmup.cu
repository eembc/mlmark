#include "hip/hip_runtime.h"
__global__ void warmup_kernel(int* c, int n) {
	int sum = 0;

#pragma unroll 1
	for (int i = 0; i < n; i++) {
		sum++;
	}
	c[0] = sum;
}


void warmupKernel(hipStream_t stream, int *c, int n)
{
	warmup_kernel << <1, 1, 0, stream >> > (c, n);
}

void checkWarmUp(hipStream_t stream, int warmUpMs)
{
	if (warmUpMs > 0)
	{
		hipEvent_t start, end;
		int *warmupBuffer = nullptr;

		unsigned int cudaEventFlags = hipEventDefault;// : hipEventBlockingSync;
		hipEventCreateWithFlags(&start, cudaEventFlags);
		hipEventCreateWithFlags(&end, cudaEventFlags);

		hipMalloc((void**)&warmupBuffer, sizeof(int));
		float totalMs = 0;

		do
		{
			hipEventRecord(start, stream);
			warmupKernel(stream, warmupBuffer, 1 << 20);
			hipEventRecord(end, stream);
			hipEventSynchronize(end);
			float ms;
			hipEventElapsedTime(&ms, start, end);
			totalMs += ms;
		} while ((int)totalMs < warmUpMs);

		hipEventDestroy(start);
		hipEventDestroy(end);
		hipFree(warmupBuffer);
	}
}