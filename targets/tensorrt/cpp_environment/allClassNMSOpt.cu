#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <vector>
#include <algorithm>
#include <stdint.h>

#include "ssdOpt.h"
#include "ssdOptMacros.h"

namespace nvinfer1
{
namespace plugin
{

namespace {
__device__ __inline__ void swap(float &a, float &b)
{
    float temp = a;
    a = temp;
    b = temp;
}

} // namespace anonymous

template <typename T_BBOX>
__device__ T_BBOX bboxSizeOpt(
    const Bbox<T_BBOX>& bbox,
    const bool normalized)
{
    if (normalized) {
        // If any bbox dimension is negative the result will be zero.
        T_BBOX width = fmaxf(bbox.xmax - bbox.xmin, 0.0f);
        T_BBOX height = fmaxf(bbox.ymax - bbox.ymin, 0.0f);
        return width * height;
    } else {
        T_BBOX width = bbox.xmax - bbox.xmin;
        T_BBOX height = bbox.ymax - bbox.ymin;
        if (width < 0 || height < 0) {
            return 0.0f;
        }
        return (width + 1.0f) * (height + 1.0f);
    }
}

template <typename T_BBOX>
__device__ void intersectBboxOpt(
    const Bbox<T_BBOX>& bbox1,
    const Bbox<T_BBOX>& bbox2,
    Bbox<T_BBOX>* intersect_bbox)
{
    intersect_bbox->xmin = max(bbox1.xmin, bbox2.xmin);
    intersect_bbox->ymin = max(bbox1.ymin, bbox2.ymin);
    intersect_bbox->xmax = min(bbox1.xmax, bbox2.xmax);
    intersect_bbox->ymax = min(bbox1.ymax, bbox2.ymax);
}

template <typename T_BBOX>
__device__ float jaccardOverlapOpt(
   const Bbox<T_BBOX>& bbox1,
   const Bbox<T_BBOX>& bbox2,
    const bool normalized)
{
    Bbox<T_BBOX> intersect_bbox;
    intersectBboxOpt(bbox1, bbox2, &intersect_bbox);

    float intersect_size = bboxSizeOpt(intersect_bbox, normalized);
    float bbox1_size = bboxSizeOpt(bbox1, normalized);
    float bbox2_size = bboxSizeOpt(bbox2, normalized);
    return intersect_size / (bbox1_size + bbox2_size - intersect_size);
}

template <typename T_BBOX>
__device__ void emptyBboxInfoOpt(
    BboxInfo<T_BBOX>* bbox_info)
{
    bbox_info->conf_score = T_BBOX(0);
    bbox_info->label = -2; // -1 is used for all labels when shared_location is ture
    bbox_info->bbox_idx = -1;
    bbox_info->kept = false;
}
/********** new NMS for only score and index array **********/


template <typename T_SCORE, typename T_BBOX, int TSIZE, bool isNormalized>
__global__ void allClassNMSOpt_kernel(
    const int num_no_use,
    const int num_classes,
    const int num_preds_per_class,
    const int top_k_,
    const float nms_threshold,
    const bool share_location,
    const bool isNormalized_unused,
    T_BBOX* bbox_data, // bbox_data should be float to preserve location information
    T_SCORE* beforeNMS_scores,
    int* beforeNMS_index_array,
    T_SCORE* afterNMS_scores,
    int* afterNMS_index_array,
    int* active_count, // number of active elemements per class/batch
    int* active_count_per_batch,
    bool flipXY = false)
{
    const int num_smem_elements = TSIZE * blockDim.x;

    // keep a small smem cache for the bboxes. Alignment is guaranteed due to the order of the definitions.
    extern __shared__ int4 dynamic_smem[];
    // number of active elements for the current batch_class combi
    __shared__ int result_active_count;

    Bbox<T_BBOX> *sh_bbox = reinterpret_cast<Bbox<T_BBOX>*>(dynamic_smem);
    bool *kept_bboxinfo_flag = reinterpret_cast<bool*>(sh_bbox + num_smem_elements);

    int active = active_count[blockIdx.y * gridDim.x + blockIdx.x];
    int top_k = (active < top_k_) ? active : top_k_;

    int class_id = blockIdx.x;
    int batch_id = blockIdx.y;

    // Each thread touches only a certain subset of all bboxinfos. Keep the kept_bboxinfo_flag for the thread in a bitmask.
    uint32_t thread_kept_bboxinfo_flag = 0;
    const int offset = batch_id * num_classes * num_preds_per_class + class_id * num_preds_per_class;

    // local thread data
    // TODO loc_bboxIndex is only required during the bbox initialization phase. don't waste registers for it...
    int loc_bboxIndex[TSIZE];
    Bbox<T_BBOX> loc_bbox[TSIZE];

    if (active)
    {
        // we do not have to synchronize after writing active_count_per_batch.
        // T_SIZE is > 0, so there'll be at least one syncthreads before the first usage of this variable.
        if (threadIdx.x == 0) {
            result_active_count = 0;
        }

        const int max_idx = offset + top_k; // put top_k bboxes into NMS calculation
        const int bbox_idx_offset = share_location ? (batch_id * num_preds_per_class) : (batch_id * num_classes * num_preds_per_class);


// {{{ initialize Bbox, Bboxinfo, kept_bboxinfo_flag
#pragma unroll
        for (int t = 0; t < TSIZE; t++)
        {
            bool thread_kept_bboxinfo = false;
            const int cur_idx = threadIdx.x + blockDim.x * t;
            const int item_idx = offset + cur_idx;

            if (item_idx < max_idx)
            {
                loc_bboxIndex[t] = beforeNMS_index_array[item_idx];
                if (loc_bboxIndex[t] != -1)
                {
                    const int bbox_data_idx = share_location ? (loc_bboxIndex[t] % num_preds_per_class + bbox_idx_offset) : loc_bboxIndex[t];
                    loc_bbox[t] = ((Bbox<T_BBOX>*)bbox_data)[bbox_data_idx];
                    if (flipXY) {
                        swap(loc_bbox[t].xmin, loc_bbox[t].ymin);
                        swap(loc_bbox[t].xmax, loc_bbox[t].ymax);
                    }
                    sh_bbox[cur_idx] = loc_bbox[t];

                    thread_kept_bboxinfo = true;
                    thread_kept_bboxinfo_flag |= (1 << t);
                }
            }
            kept_bboxinfo_flag[cur_idx] = thread_kept_bboxinfo;
        }

        // }}}
        __syncthreads();

        // TODO we can use loc_bboxIndex[t] == -1 to find the maximum index which is -1 and set max_idx to this value. This would reduce
        // the number of iterations for all threads if there are less than top-k bboxes available. How likey is this?

        // {{{ filter out overlapped boxes with lower scores
        {
            const int offset = 0;
            const int max_idx = top_k;
            int ref_item_idx = 0;

            while (ref_item_idx < max_idx)
            {
                Bbox<T_BBOX> ref_bbox;
                //*((int4*)&ref_bbox) = *((int4*)&sh_bbox[ref_item_idx - offset]);
                ref_bbox = sh_bbox[ref_item_idx];

                //uint32_t enabled = ~1;
                for (int t = 0; t < TSIZE; t++)
                {
                    const int cur_idx = threadIdx.x + blockDim.x * t;
                    const int item_idx = offset + cur_idx;

                    if ((item_idx > ref_item_idx) && (thread_kept_bboxinfo_flag & (1 << t)))
                    {
                        if (jaccardOverlapOpt(ref_bbox, loc_bbox[t], isNormalized) > nms_threshold)
                        {
                            thread_kept_bboxinfo_flag &= ~(1 << t);
                            kept_bboxinfo_flag[cur_idx] = false;
                        }
                    }
                }
                __syncthreads();

                do
                {
                    ref_item_idx++;
                } while (ref_item_idx < max_idx && !kept_bboxinfo_flag[ref_item_idx - offset]);
            }
        }
        // }}}

        // {{{ store data
        // Ideally we'd compact the data for the next stage to reduce work on the next stage.
        // As long as there's no TopK algorithm with a dynamic number of elements for the input
        // it doesn't make sense yet to do the compact step.

        // first determine the total amount of active elements after the NMS step
        int thread_active =  __popc(thread_kept_bboxinfo_flag);
        if (thread_active) {
            int write_offset = atomicAdd(&active_count_per_batch[batch_id], thread_active);
            int write_item_idx = (batch_id * num_classes * top_k_) + write_offset;
            for (int t = 0; t < TSIZE; t++) {
                const int cur_idx = threadIdx.x + blockDim.x * t;
                const int read_item_idx = offset + cur_idx;

                bool is_valid_bbox = (thread_kept_bboxinfo_flag & (1 << t));
                if (is_valid_bbox) {
                    afterNMS_scores[write_item_idx] = beforeNMS_scores[read_item_idx];
                    afterNMS_index_array[write_item_idx] = loc_bboxIndex[t];
                    ++write_item_idx;
                }
            }
        }
        // }}}
    }
}

template <typename T_SCORE, typename T_BBOX>
ssdStatus_t allClassNMSOpt_gpu(
    hipStream_t stream,
    const int num,
    const int num_classes,
    const int num_preds_per_class,
    const int top_k,
    const float nms_threshold,
    const bool share_location,
    const bool isNormalized,
    void* bbox_data,
    void* beforeNMS_scores,
    void* beforeNMS_index_array,
    void* afterNMS_scores,
    void* afterNMS_index_array,
    void* active_count,
    void* active_count_per_batch,
    bool flipXY = false)
{
#define NMS_P(tsize) allClassNMSOpt_kernel<T_SCORE, T_BBOX, (tsize), true>
#define NMS_P_U(tsize) allClassNMSOpt_kernel<T_SCORE, T_BBOX, (tsize), false>

    void (*kernel[2][8])(const int, const int, const int, const int, const float,
                         const bool, const bool, float*, T_SCORE*, int*, T_SCORE*, int*,
                         int*, int*, bool)
        = {
        {NMS_P_U(1), NMS_P_U(2), NMS_P_U(3), NMS_P_U(4), NMS_P_U(5), NMS_P_U(6), NMS_P_U(7), NMS_P_U(8),},
        {NMS_P(1), NMS_P(2), NMS_P(3), NMS_P(4), NMS_P(5), NMS_P(6), NMS_P(7), NMS_P(8),}
        };

    // round up #threads to the minimum cta size possible which holds 1 bbox per thread
    // TODO 1024 is the #threads per CTA limit and should be queried from the GPU.
    // With top_k > max #threads per CTA this heuristic gets inefficient and should be enhanced
    // to reduce the number of idle threads.
    const int BS = std::min(((top_k + 31) / 32) * 32, 1024);
    const dim3 GS(num_classes, num);
    const int t_size = (top_k + BS - 1) / BS;
    assert(t_size < 8);

    // compute smem size for bbox cache and kept boxes
    const int smem_size = BS * t_size * (sizeof(bool) + sizeof(Bbox<T_BBOX>));
    kernel[isNormalized][t_size - 1]<<<GS, BS, smem_size, stream>>>(num, num_classes, num_preds_per_class,
                                                                    top_k, nms_threshold, share_location, isNormalized,
                                                                    (T_BBOX*) bbox_data,
                                                                    (T_SCORE*) beforeNMS_scores,
                                                                    (int*) beforeNMS_index_array,
                                                                    (T_SCORE*) afterNMS_scores,
                                                                    (int*) afterNMS_index_array,
                                                                    (int*) active_count,
                                                                    (int*) active_count_per_batch,
                                                                    flipXY);

    CSC(hipGetLastError(), STATUS_FAILURE);
    return STATUS_SUCCESS;
}

// allClassNMSOpt LAUNCH CONFIG {{{
typedef ssdStatus_t (*nmsOptFunc)(hipStream_t,
                               const int,
                               const int,
                               const int,
                               const int,
                               const float,
                               const bool,
                               const bool,
                               void*,
                               void*,
                               void*,
                               void*,
                               void*,
                                  void*, // activeCount
                                  void*, // activeCountPerClass
                               bool);

struct nmsOptLaunchConfigSSD
{
    DType_t t_score;
    DType_t t_bbox;
    nmsOptFunc function;

    nmsOptLaunchConfigSSD(DType_t t_score, DType_t t_bbox)
        : t_score(t_score)
        , t_bbox(t_bbox)
    {
    }
    nmsOptLaunchConfigSSD(DType_t t_score, DType_t t_bbox, nmsOptFunc function)
        : t_score(t_score)
        , t_bbox(t_bbox)
        , function(function)
    {
    }
    bool operator==(const nmsOptLaunchConfigSSD& other)
    {
        return t_score == other.t_score && t_bbox == other.t_bbox;
    }
};

static std::vector<nmsOptLaunchConfigSSD> nmsOptFuncVec;

bool nmsOptInit()
{
    nmsOptFuncVec.push_back(nmsOptLaunchConfigSSD(DataType::kFLOAT, DataType::kFLOAT,
                                            allClassNMSOpt_gpu<float, float>));
    return true;
}

static bool initialized = nmsOptInit();

//}}}

ssdStatus_t allClassNMSOpt(hipStream_t stream,
                           const int num,
                           const int num_classes,
                           const int num_preds_per_class,
                           const int top_k,
                           const float nms_threshold,
                           const bool share_location,
                           const bool isNormalized,
                           const DType_t DT_SCORE,
                           const DType_t DT_BBOX,
                           void* bbox_data,
                           void* beforeNMS_scores,
                           void* beforeNMS_index_array,
                           void* afterNMS_scores,
                           void* afterNMS_index_array,
                           void* active_count,
                           void* active_count_per_batch,
                           bool flipXY)
{
    nmsOptLaunchConfigSSD lc = nmsOptLaunchConfigSSD(DT_SCORE, DT_BBOX, allClassNMSOpt_gpu<float, float>);
    for (unsigned i = 0; i < nmsOptFuncVec.size(); ++i)
    {
        if (lc == nmsOptFuncVec[i])
        {
            DEBUG_PRINTF("all class nms kernel %d\n", i);
            return nmsOptFuncVec[i].function(stream,
                                          num,
                                          num_classes,
                                          num_preds_per_class,
                                          top_k,
                                          nms_threshold,
                                          share_location,
                                          isNormalized,
                                          bbox_data,
                                          beforeNMS_scores,
                                          beforeNMS_index_array,
                                          afterNMS_scores,
                                          afterNMS_index_array,
                                          active_count,
                                          active_count_per_batch,
                                          flipXY);
        }
    }
    return STATUS_BAD_PARAM;
}

} // namespace plugin
} // namespace nvinfer1
